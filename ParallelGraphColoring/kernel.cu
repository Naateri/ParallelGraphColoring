#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hipsparse.h"

#include <stdio.h>
#include <iostream>
#include <fstream>

#include <stdlib.h>
#include <time.h>
#include <algorithm>

#include <thrust/count.h>

#include <random>
// random number generator
#include <intrin.h>

#pragma intrinsic(__rdtsc)

using namespace std;

// Reads Matrix in Matrix Market format (.mtx)
// Returns Matrix in Compressed-Row Format (CSR)
void read_mat(char* filename, float*& csrValA, int*& csrRowPtrA, int*& csrColIndA, int& cols, int& rows, int& nnz) {

    cout << "Reading matrix " << filename << endl;

    ifstream readfile(filename);

    while (readfile.peek() == '%') readfile.ignore(2048, '\n');
    /*
    std::string str;
    getline(readfile, str);
    char c;
    sscanf(str.c_str(), "%c", &c);
    while (c == '%') {
        getline(readfile, str);
        sscanf(str.c_str(), "%c", &c);
    }
    */
    // Ignore lines with % (comment)
    // First line: dimension of matrix + non-zero values
    // Read defining parameters:
    readfile >> rows >> cols >> nnz;

    csrValA = new float[nnz];
    csrRowPtrA = new int[rows + 1]; csrRowPtrA[0] = 0;
    csrColIndA = new int[nnz];

    int* row_amount = new int[rows];
    // init with zeros

    for (int row = 0; row < rows; row++) row_amount[row] = 0;

    // Second and beyond lines: row column value

    for (int i = 0; i < nnz; i++)
    {
        int m, n;
        float data;
        readfile >> m >> n >> data;
        row_amount[m - 1]++;
        csrValA[i] = data;
        csrColIndA[i] = n - 1;
    }

    int count = 0;
    // Finding cumulative sum
    /*for (int row = 0; row < rows; row++) {
        csrRowPtrA[row] += count;
        count += row_amount[row];
    }
    csrRowPtrA[rows] += count;*/

    for (int row = 1; row <= rows; row++) {
        csrRowPtrA[row] = csrRowPtrA[row - 1] + row_amount[row - 1];
    }

    readfile.close();

    delete[] row_amount;

    cout << "Matrix stored in CSR format.\n";
}

// graph coloring

__global__ void color_jpl_kernel(int n, int c, const int* Ao,
    const int* Ac, const float* Av,
    const int* randoms, int* colors)
{
   for (int i = threadIdx.x + blockIdx.x * blockDim.x;
        i < n;
        i += blockDim.x * gridDim.x)
    {
    //int i = threadIdx.x + blockIdx.x * blockDim.x;
    //if (i < n){
        bool f = true; // true iff you have max random

        // ignore nodes colored earlier
        if ((colors[i] != -1)) continue;
        //if ((colors[i] != -1)) return;

        int ir = randoms[i];

        // look at neighbors to check their random number
        for (int k = Ao[i]; k < Ao[i + 1]; k++) {
            // ignore nodes colored earlier (and yourself)
            int j = Ac[k];
            int jc = colors[j];
            if (((jc != -1) && (jc != c)) || (i == j)) continue;
            int jr = randoms[j];
            if (ir <= jr) {
                f = false;
                break;
            }
        }

        // assign color if you have the maximum random number
        if (f) colors[i] = c;
    }
}

int get_rand(int max) {
    srand((unsigned)time(NULL));
    //srand(__rdtsc());
    return rand() % max;

   /* string str = "test";
    std::seed_seq seed1(str.begin(), str.end());

    std::mt19937 g2(seed1);

    return (int)g2();*/
}

void init_rand_array(int*& randoms, int size) {
    for (int i = 0; i < size; i++) {
        randoms[i] = get_rand(size << 2);
        //cout << "rand " << randoms[i] << endl;
    }
}

void color_jpl(int n,
    const int* Ao, const int* Ac, const float* Av,
    int* colors, int* d_randoms)
{

    thrust::fill(colors, colors + n, -1); // init colors to -1

    int* d_colors;
    hipMalloc((void**)&d_colors, n * sizeof(int));
    hipMemcpy(d_colors, colors, n * sizeof(int), hipMemcpyHostToDevice);

    cout << "initiallized random numbers and colors\n";

    cout << "nodes left: " << (int)thrust::count(colors, colors + n, -1) << endl;
    for (int c = 0; c < n; c++) {
        int nt = 256;
        //int nb = min((n + nt - 1) / nt, 1000);
        int nb = (ceil(n / nt));
        //cout << "color: " << c << endl;
        color_jpl_kernel << <nb, nt >> > (n, c,
            Ao, Ac, Av,
            d_randoms,
            d_colors);
        hipDeviceSynchronize();
        hipMemcpy(colors, d_colors, n * sizeof(int), hipMemcpyDeviceToHost);
        int left = (int)thrust::count(colors, colors + n, -1);
        cout << "nodes left: " << left << endl;
        if (left == 0) break;
    }

    hipDeviceSynchronize();

    hipMemcpy(colors, d_colors, n * sizeof(int), hipMemcpyDeviceToHost);

    //delete[] randoms;
    //hipFree(d_randoms);
    hipFree(d_colors);
}



int main()
{
    float* csrVal;
    int* csrRowPtr;
    int* csrColInd;

    int nnz, rows, cols;
    read_mat("Matrices/offshore.mtx", csrVal, csrRowPtr, csrColInd, cols, rows, nnz);
    //read_mat("Matrices/parabolic_fem.mtx", csrVal, csrRowPtr, csrColInd, cols, rows, nnz);

    cout << "Rows cols nnz " << rows << " " << cols << " " << nnz << endl;

    int* d_csrRowPtr, * d_csrColInd;
    float* d_csrVal;

    // Separating space on GPU for matrix
    hipMalloc((void**)&d_csrRowPtr, (rows + 1) * sizeof(int));
    hipMalloc((void**)&d_csrColInd, nnz * sizeof(int));
    hipMalloc((void**)&d_csrVal, nnz * sizeof(float));

    // color and reordering info

    int ncolors = 0, * coloring;
    int* d_coloring, * d_reordering;
    float fraction = 1.0;
    coloring = (int*)calloc(rows, sizeof(int));

    // separating space for colors and reordering in gpu

    hipMalloc((void**)&d_coloring, rows * sizeof(int));
    hipMalloc((void**)&d_reordering, rows * sizeof(int));
    hipMemset(d_reordering, 0, rows * sizeof(int));
    hipDeviceSynchronize();

    // Sending matrix info to GPU

    hipMemcpy(d_csrRowPtr, csrRowPtr, (rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColInd, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrVal, csrVal, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipsparseStatus_t status;
    hipsparseHandle_t handle;
    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("error!");
        exit(1);
    }
    hipsparseMatDescr_t descr;
    status = hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("error!");
        exit(1);
    }

    hipsparseColorInfo_t info;
    status = hipsparseCreateColorInfo(&info);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("error!");
        exit(1);
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // color
    status = hipsparseScsrcolor(handle, rows, nnz, descr, d_csrVal, d_csrRowPtr, d_csrColInd, &fraction, &ncolors, d_coloring, d_reordering, info);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Milliseconds for operation (CC): " << milliseconds << endl;
    cout << "Colors: " << ncolors << endl;
    
    /*
    for (int i = 0; i < rows; i++) {
        printf("coloring[%d]: %d\n", i, coloring[i]);
    }

    for (int i = 0; i < rows; i++) {
        printf("reordering[%d]: %d\n", i, reordering[i]);
    }*/
    //cout << "coloring " << coloring << endl;
    //cout << "reordering " << reordering << endl;

    int* colors = new int[rows];
    //int* colors;
    //hipMallocManaged((void**)&colors, nnz * sizeof(int));
    //int* d_colors;
    //hipMalloc((void**)&d_colors, nnz * sizeof(int));
    cout << "JPL algorithm time\n";
    
    int* randoms; // allocate and init random array 
    randoms = new int[rows];

    cout << "Initializing random values\n";
    init_rand_array(randoms, rows);
    cout << "Rand values initialized\n";

    int* d_randoms;
    hipMalloc((void**)&d_randoms, rows * sizeof(int));

    hipMemcpy(d_randoms, randoms, rows * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    hipEventRecord(start2);
    color_jpl(rows, d_csrRowPtr, d_csrColInd, d_csrVal, colors, d_randoms);
    hipEventRecord(stop2);

    hipEventSynchronize(stop2);
    float jpl_milli = 0;
    hipEventElapsedTime(&jpl_milli, start2, stop2);

    //hipMemcpy(colors, d_colors, nnz * sizeof(int), hipMemcpyDeviceToHost);

    cout << "Milliseconds for operation (JPL): " << jpl_milli << endl;

    int jpl_colors = 0;
    for (int i = 0; i < rows; i++) {
        if (colors[i] > jpl_colors) {
            jpl_colors = colors[i];
        }
    }

    cout << "Colors: " << jpl_colors << endl;
    
    delete[] csrVal;
    delete[] csrRowPtr;
    delete[] csrColInd;
    delete[] colors;

    hipFree(d_csrRowPtr);
    hipFree(d_csrColInd);
    hipFree(d_csrVal);

}